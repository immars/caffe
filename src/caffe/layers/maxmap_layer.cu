#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxMapForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int cur_num = index / channels;
    int cur_channel = index % channels;
    const Dtype* cur_bottom = bottom_data +
        (cur_num * channels + cur_channel) * height * width;
    Dtype* cur_top = top_data + 2*index;
    int pixel = 0;
    Dtype max = -FLT_MAX;
    for (int h = 0; h < height; h++) {
      for (int w = 0; w < width; w++) {
        if (cur_bottom[pixel] > max) {
          max = cur_bottom[pixel];
          cur_top[0] = h;
          cur_top[1] = w;
        }
        pixel++;
      }
    }
  }
}


template <typename Dtype>
void MaxMapLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
//  this->Forward_cpu(bottom, top);

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count()/2;
  // NOLINT_NEXT_LINE(whitespace/operators)
  MaxMapForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->num(), bottom[0]->channels(),
      bottom[0]->height(), bottom[0]->width(), top_data);

  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void MaxMapLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  // Do not backward
  caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff);
}

#ifdef CPU_ONLY
STUB_GPU(MaxMapLayer);
#endif

INSTANTIATE_LAYER_GPU_FUNCS(MaxMapLayer);

}  // namespace caffe
